#include "hip/hip_runtime.h"
#include <ctime>
#include <fstream>
#include <iostream>

#include "camera.h"
#include "hittable_list.h"
#include "material.h"
#include "sphere.h"


#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const* const func, const char* const file,
                int const line) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":"
              << line << " '" << func << "' \n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}

void writeToFile(int image_width, int image_height, vec3* fb, size_t s) {
  std::ofstream output("image.ppm");
  output << "P3\n" << image_width << ' ' << image_height << "\n255\n";
  for (size_t i = 0; i < image_width * image_height; ++i){
    output << int(255.99 * fb[i].x()) << ' ' << int(255.99 * fb[i].y()) << ' ' << int(255.99*fb[i].z()) << '\n';
  }
}

__global__ void render(vec3* fb, int image_width, int image_height) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= image_width) || (j >= image_height)) return;
    int idx = j * image_width + i;

    fb[idx] = vec3(float(i) / (image_width - 1), float(j) / (image_height - 1), 0.0f);
}

int main() {

  // Image + CUDA buffer

  int image_width = 1200;
  int image_height = 600;

  int num_pixels = image_width * image_height;
  size_t fb_size = num_pixels * sizeof(vec3);
  vec3* fb;
  checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

  int t_width = 8;
  int t_height = 8;

  dim3 blocks(image_width / t_width + 1, image_height / t_height + 1);
  dim3 threads(t_width, t_height);

  // Render
  clock_t start, stop;
  start = clock();
  render<<<blocks, threads>>>(fb, image_width, image_height);
  checkCudaErrors(hipDeviceSynchronize()); // Blocks until all threads have completed.
  stop = clock();
  std::cout << (double)(stop - start) / CLOCKS_PER_SEC << '\n';

  writeToFile(image_width, image_height, fb, fb_size);
  checkCudaErrors(hipFree(fb));
}
